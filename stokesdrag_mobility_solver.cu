// stokesdrag_mobility_solver.cu

#include <iomanip>
#include <iostream>
#include <cmath>
#include <algorithm>
#include "stokesdrag_mobility_solver.hpp"
#include "cuda_functions.hpp"
#include "swimmer.hpp"

stokesdrag_mobility_solver::~stokesdrag_mobility_solver(){}

stokesdrag_mobility_solver::stokesdrag_mobility_solver(){}

void stokesdrag_mobility_solver::evaluate_segment_segment_mobility(){

  int start_seg = 0;

    for (int n = 0; n < num_gpus; n++){

      hipSetDevice(n);

      int num_thread_blocks = (num_segs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

      Ms_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_segs_device[n], f_segs_device[n], start_seg, num_segs[n]);

      start_seg += num_segs[n];

    }

}

void stokesdrag_mobility_solver::evaluate_segment_blob_mobility(){

  return;

}

void stokesdrag_mobility_solver::evaluate_blob_blob_mobility(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    int num_thread_blocks = (num_blobs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Mb_mult<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_blobs_device[n], f_blobs_device[n], start_blob, num_blobs[n]);

    start_blob += num_blobs[n];

  }

}

void stokesdrag_mobility_solver::evaluate_blob_segment_mobility(){

  int start_blob = 0;

  for (int n = 0; n < num_gpus; n++){

    hipSetDevice(n);

    const int num_thread_blocks = (num_blobs[n] + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    Mb_fill_zero<<<num_thread_blocks, THREADS_PER_BLOCK>>>(v_blobs_device[n], start_blob, num_blobs[n]);

    start_blob += num_blobs[n];

  }

}

